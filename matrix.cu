#include "matlib.h"
#include "file/muhfile.h"


int main(int argc, char const *argv[]){

	// dimensions for matrices. m*n . n*k
	int m, n, k;
	srand(time(NULL));
	int dec = 0;
	float elapsed_time;
	FILE *fp;// file pointer, used for both in and out since they dont overlap
	char filename[BUFFSIZE];// storing filename
	char *p; // removing trailing \n in filename input
	char *fileerror;

	int *host_a, *host_b, *host_c;	//  host array pointers
	int *dev_a, *dev_b, *dev_c;		//	device array pointers

	printf("Would you like to load your arrays from a file(1) or have them");
	printf(" generated(0)? ");
	dec = getchar() - '0';

	if(dec){ // loads from file
		while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff
		//char buff[BUFFSIZE];	// for reading in lines to be parsed

		printf("Enter filename for input: ");
		fgets(filename, BUFFSIZE ,stdin);				// read in filename
		//FILENAME FIXER
        if((p = strchr(filename, '\n')) != NULL){		// fix filename
			//printf("in the filenam fixer\n");
            *p = '\0';								
        }

		printf("Opening \"%s\" for reading\n", filename);	//open filename
		fp = fopen(filename,"r"); 

		//GET MATRICES DIMENSIONS
	    fscanf(fp, "%d %d %d", &m, &n, &k);
//		newline();

//CONSOLIDATE THESE MALLOCS TO BEFORE THE IF ELSE IF POSSIBLE

		hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
		hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // first array

		//	fille matricies from file
		fillmatfromfile(fp, host_a, m, n);
		fillmatfromfile(fp, host_b, n, k);
		
		if(!fclose(fp)){
			printf("Closed \"%s\"\n", filename);
		}else{
			printf("No Close for YOU\n");
		}
		
	}else{							// randomly generates matricies
		while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff
	    printf("Would you you like to save the randomly generated matrices");
	    printf(" to a file (y/n)? ");
		
		dec = getchar();

		// *** can you clean up the flow? this is in both if and else. 
		if(dec == 'y'){			// saves the randome matricies to a file

			while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff XXXXXXXXXX

			printf("Enter filename for saving: ");
			fileerror = fgets(filename, BUFFSIZE ,stdin);
			if(fileerror == NULL){
				printf("Could not open \"%s\"\n", filename);
			}
			// filename fixer
			if((p = strchr(filename, '\n')) != NULL){ //remove newline
				*p = '\0';								
			}

			if(fp = fopen(filename,"w")){
				printf("Opened \"%s\" for writing\n", filename);
			}else{
				printf("Could not open \"%s\" for writing\n", filename);
			}	
	
		    printf("please type in array dimensions m, n, and k: ");
		    scanf("%d %d %d", &m, &n, &k);
// YOU NEED TO  MOVE THIS TOBEFORE THE IF ELSE STATEMENT!!! THIS IS NOT CLEAN!!
			hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
			hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // second array
			hipHostMalloc((void **) &host_c, sizeof(int) * m * k); // product

//			while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff XXXXXXXXXX ?????????

			// fills Matrices and stores them in a file
			matfilefill(fp, host_a, host_b, m, n, k);
			printf("Matrices filled and output to file \"%s\"\n", filename);

			if(!fclose(fp)){
				printf("Closed \"%s\"\n", filename);
			}else{
				printf("lol no close for you\n");
			}

		}else{ //could check if n but... well... get to that later...

		    printf("please type in array dimensions m, n, and k: ");
		    scanf("%d %d %d", &m, &n, &k);
// YOU NEED TO  MOVE THIS TOBEFORE THE IF ELSE STATEMENT!!! THIS IS NOT CLEAN!!
			hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
			hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // second array
			hipHostMalloc((void **) &host_c, sizeof(int) * m * k); // product
			matfill(host_a, m, n);
			matfill(host_b, n, k);
		}

	}

	// PRINT
	matprint(host_a, m, n);
	printf("\n");
	matprint(host_b, n, k);
	printf("\n");

//printf("OOT OF THE IF?ELSE\n");

	// for measuring time	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start timer
	hipEventRecord(start, 0);
	
	
	// Allocate space for arrays on DEVICE
	hipMalloc((void **) &dev_a, sizeof(int) * m * n); // first array
	hipMalloc((void **) &dev_b, sizeof(int) * n * k); // first array
	hipMalloc((void **) &dev_c, sizeof(int) * m * k); // product of arrays

	// Copy matricies from HOST to DEVICE
	hipMemcpy(dev_a, host_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

	// Preparind dimGrid and dimBlock for use in gpu_matrix_mult function
	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE; 
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE; 
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// launch that fam
	gpu_matrix_mult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, m, n, k);

	// transfer results to host
	hipMemcpy(host_c, dev_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);

	// end timer
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	
//	printf("Matricies multiplied: %d * %d . %d * %d\n", m, n, ,n, k);
//	printf("abbagamba\n");

	printf("World Domination Computation Completed Professor\n");
	matprint(host_c, m, k);
	printf("Time elapsed: %f\n", elapsed_time);

	// Free the mmeory so it isnt banished into eternity
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
//	printf("cudaFreed\n");
	hipHostFree(host_a);
//	printf("cudaFreedHost\n");
	hipHostFree(host_b);
	hipHostFree(host_c);

	printf("Program OVER!\n");
}
