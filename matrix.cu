#include "matlib.h"
#include "file/muhfile.h"


int main(int argc, char const *argv[]){

	// dimensions for matrices. m*n . n*k
	int m, n, k;
	srand(time(NULL));
	int dec = 0;
	float elapsed_time;
	FILE *fp;// file pointer, used for both in and out since they dont overlap
	char filename[BUFFSIZE];// storing filename
	char *p; // removing trailing \n in filename input
	char *fileerror;

	
	int *host_a, *host_b, *host_c;	//  host array pointers
	int *dev_a, *dev_b, *dev_c;		//	device array pointers

	printf("Would you like to load your arrays from a file(1) or have them");
	printf(" generated(0)? ");
	dec = getchar() - '0';

//	printf("dec = %d", dec);
	newline();

	if(dec){
		while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff
		//char buff[BUFFSIZE];	// for reading in lines to be parsed

		printf("Enter filename for input: ");
		fgets(filename, BUFFSIZE ,stdin);				// read in filename
		//FILENAME FIXER
        if((p = strchr(filename, '\n')) != NULL){		// fix filename
			printf("in the filenam fixer\n");
            *p = '\0';								
        }

		printf("Opening %s for reading\n", filename);	//open filename
		fp = fopen(filename,"r"); 

		//GET MATRICES DIMENSIONS
	    fscanf(fp, "%d %d %d", &m, &n, &k);
		newline();

//CONSOLIDATE THESE MALLOCS TO BEFORE THE IF ELSE IF POSSIBLE

		hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
		hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // first array

		//	fille matricies from file
		fillmatfromfile(fp, host_a, m, n);
		fillmatfromfile(fp, host_b, n, k);
		
		if(!fclose(fp)){
			printf("CLOsinG TiME\n");
		}else{
			printf("No Close for YOU\n");
		}
		
	}else{
		while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff
	    printf("Would you you like to save the randomly generated matrices");
	    printf(" to a file (y/n)? ");
		
		dec = getchar();
		//printf("%c\n", (char)dec);

		// *** can you clean up the flow? this is in both if and else. 
		if(dec == 'y'){

			printf("dec = %d\n", (int)dec);
			while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff XXXXXXXXXX
			printf("dec = %d\n", (int)dec);

			printf("Enter filename for saving: ");
			fileerror = fgets(filename, BUFFSIZE ,stdin);
			if(fileerror == NULL){
				printf("Could not open %s\n", filename);
			}
			//FILENAME FIXER
			if((p = strchr(filename, '\n')) != NULL){ //remove newline
//				printf("in the filename fixer\n");
				*p = '\0';								
			}

			printf("Opening %s for writing\n", filename);
			if(fp = fopen(filename,"w")){
				printf("Opened %s for writing\n", filename);
			}else{
				printf("Could not open %s for writing\n", filename);
			}	
	
		    printf("please type in array dimensions m, n, and k: ");
		    scanf("%d %d %d", &m, &n, &k);

// YOU NEED TO  MOVE THIS TOBEFORE THE IF ELSE STATEMENT!!! THIS IS NOT CLEAN!!
			hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
			hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // second array
			hipHostMalloc((void **) &host_c, sizeof(int) * m * k); // product

// CHANGE VARIABLe; Clean this up
			while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff XXXXXXXXXX
		// FILL MATRICES AND FILE
/*


			printf("Enter filename for saving: ");
			fileerror = fgets(filename, BUFFSIZE ,stdin);
			if(fileerror == NULL){
				printf("Could not open %s\n", filename);
			}
			//FILENAME FIXER
			if((p = strchr(filename, '\n')) != NULL){ //remove newline
//				printf("in the filename fixer\n");
				*p = '\0';								
			}

			printf("Opening %s for writing\n", filename);
			if(fp = fopen(filename,"w")){
				printf("Opened %s for writing\n", filename);
			}else{
				printf("Could not open %s for writing\n", filename);
			}	
	


*/
		// FILLs Matrices and stores them in a file
			matfilefill(fp, host_a, host_b, m, n, k);
			printf("Matrices filled and output to file %s\n", filename);

			if(!fclose(fp)){
				printf("Closed %s\n", filename);
			}else{
				printf("lol no close for you\n");
			}









		}else{ // could check if n but... well... get to that later...
		    printf("please type in array dimensions m, n, and k: ");
		    scanf("%d %d %d", &m, &n, &k);
// YOU NEED TO  MOVE THIS TOBEFORE THE IF ELSE STATEMENT!!! THIS IS NOT CLEAN!!
			hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
			hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // second array
			hipHostMalloc((void **) &host_c, sizeof(int) * m * k); // product
			matfill(host_a, m, n);
			matfill(host_b, n, k);
		}

		// PRINT
			matprint(host_a, m, n);
			printf("\n");
			matprint(host_b, n, k);
			printf("\n");
	}

printf("OOT OF THE IF?ELSE\n");

	// for measuring time	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start timer
	hipEventRecord(start, 0);
	
	
	// Allocate space for arrays on DEVICE
	hipMalloc((void **) &dev_a, sizeof(int) * m * n); // first array
	hipMalloc((void **) &dev_b, sizeof(int) * n * k); // first array
	hipMalloc((void **) &dev_c, sizeof(int) * m * k); // product of arrays

	// Copy matricies from HOST to DEVICE
	hipMemcpy(dev_a, host_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

	// Preparind dimGrid and dimBlock for use in gpu_matrix_mult function
	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE; 
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE; 
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// launch that fam
	gpu_matrix_mult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, m, n, k);

	// transfer results to host
	hipMemcpy(host_c, dev_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);

	// end timer
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	
//	printf("Matricies multiplied: %d * %d . %d * %d\n", m, n, ,n, k);
//	printf("abbagamba\n");

	printf("World Domination Computation Completed Professor\n");
	matprint(host_c, m, k);
	printf("Time elapsed: %f\n", elapsed_time);

	// Free the mmeory so it isnt banished into eternity
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	printf("cudaFreed\n");
	hipHostFree(host_a);
	printf("cudaFreedHost\n");
	hipHostFree(host_b);
	hipHostFree(host_c);

	printf("Program OVER!\n");
}
