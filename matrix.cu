#include "matlib.h"
#include "file/muhfile.h"


int main(int argc, char const *argv[]){

	// dimensions for matrices. m*n . n*k
	int m, n, k;
	srand(time(NULL));
	int dec = 0;
	float elapsed_time;
	FILE *fp;// file pointer, used for both in and out since they dont overlap
	char filename[BUFFSIZE];// storing filename
	char *p; // removing trailing \n in filename input
	char *fileerror;

	
	int *host_a, *host_b, *host_c;	//  host array pointers
	int *dev_a, *dev_b, *dev_c;		//	device array pointers

	printf("Would you like to load your arrays from a file(1) or have them");
	printf(" generated(0)? ");
	dec = getchar() - '0';
	printf("dec = %d", dec);

	printf("\n");

	if(dec){
		while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff
		//char buff[BUFFSIZE];	// for reading in lines to be parsed

		printf("Enter filename for input: ");
		fgets(filename, BUFFSIZE ,stdin);				// read in filename
		//FILENAME FIXER
        if((p = strchr(filename, '\n')) != NULL){		// fix filename
			printf("in the filenam fixer\n");
            *p = '\0';								
        }

		printf("Opening %s for reading\n", filename);	//open filename
		fp = fopen(filename,"r"); 

//;asdilf;lasdfk;sdfjlak;sdfjlalk;asdfjasdfkl;jasdfkl;j





		printf("before the while\n");

		//GET MATRICES DIMENSIONS
	    fscanf(fp, "%d %d %d", &m, &n, &k);
		newline();


//CONSOLIDATE THESE MALLOCS TO BEFORE THE IF ELSE IF POSSIBLE
		hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
		hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // first array

		fillmatfromfile(fp, host_a, m, n);
		fillmatfromfile(fp, host_b, n, k);
		
		printf("CLOsinG TiME\n");
		fclose(fp);


/* 
THIS IS THE PARTS YOURE WORKING ON
read in the matrices and store them in the arrays
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
*/

	}else{

//	    printf("Would you you like to save the randomly generated matrices");
//	    printf(" to a file?\n");
//		fgets(filename, BUFFSIZE ,stdin);

/*
	1. store randomly generated matricies in a file
	2. 
	3. add control structure
*/
		// *** can you clean up the flow? this is in both if and else. 

	    printf("please type in array dimensions m, n, and k: ");
	    scanf("%d %d %d", &m, &n, &k);

// YOU NEED TO  MOVE THIS TOBEFORE THE IF ELSE STATEMENT!!! THIS IS NOT CLEAN!!

		hipHostMalloc((void **) &host_a, sizeof(int) * m * n); // first array
		hipHostMalloc((void **) &host_b, sizeof(int) * n * k); // second array
		hipHostMalloc((void **) &host_c, sizeof(int) * m * k); // product

// CHANGE VARIABLe; Clean this up
		while((dec = getchar()) != '\n' && dec != EOF){} // for clearing inbuff
	
		// FILL MATRICES AND FILE

			printf("Enter filename for saving: ");
			fileerror = fgets(filename, BUFFSIZE ,stdin);
			if(fileerror == NULL){
				printf("fileerror = NULL; YHOU FUHAILED!\n");
			}
			//FILENAME FIXER
			if((p = strchr(filename, '\n')) != NULL){ //remove newline
				printf("in the filenam fixer\n");
				*p = '\0';								
			}

			printf("Opening %s for writing\n", filename);
			fp = fopen(filename,"w");
			printf("Opened %s for writing\n", filename);
		
	// FILLs Matrices and stores them in a file
			matfilefill(fp, host_a, host_b, m, n, k);
			printf("Matrices filled and output to file %s\n", filename);
			fclose(fp);


		// PRINT
		matprint(host_a, m, n);
		printf("\n");
		matprint(host_b, n, k);
		printf("\n");
/*
		if(matprint(host_a, m, n)){
			printf("Matrix A filled\n");
		}else{
			printf("DANGER DANGER!!!! NO MATRIX IN THE FLUX RAY!!\n");
		}

		if(matprint(host_b, n, k)){
			printf("Matrix B filled\n");
		
		}else{
			printf("DANGER DANGER!!!! NO MATRIX IN THE QUANTUM CARBURETOR!!\n");
		}
*/
	}

printf("OOT OF THE IF?ELSE\n");

	//SEG FAULT!! YOU NEED TO GO BACK TO THE : BLURG



	// for measuring time	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start timer
	hipEventRecord(start, 0);
	
	
	// Allocate space for arrays on DEVICE
	hipMalloc((void **) &dev_a, sizeof(int) * m * n); // first array
	hipMalloc((void **) &dev_b, sizeof(int) * n * k); // second array
	hipMalloc((void **) &dev_c, sizeof(int) * m * k); // product of arrays

	// Copy matricies from HOST to DEVICE
	hipMemcpy(dev_a, host_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

	// Preparind dimGrid and dimBlock for use in gpu_matrix_mult function
	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE; 
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE; 
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

	// launch that fam
	gpu_matrix_mult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, m, n, k);

	// transfer results to host
	hipMemcpy(host_c, dev_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);

	// end timer
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	
//	printf("Matricies multiplied: %d * %d . %d * %d\n", m, n, ,n, k);
//	printf("abbagamba\n");

	printf("World Domination Computation Completed Professor\n");
	matprint(host_c, m, k);
	printf("Time elapsed: %f\n", elapsed_time);

	// Free the mmeory so it isnt banished into eternity
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	printf("cudaFreed\n");
	hipHostFree(host_a);
	printf("cudaFreedHost\n");
	hipHostFree(host_b);
	hipHostFree(host_c);

	printf("Program OVER!\n");
}
